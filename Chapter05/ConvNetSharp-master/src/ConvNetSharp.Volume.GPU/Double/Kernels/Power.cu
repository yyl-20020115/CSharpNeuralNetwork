
#include <hip/hip_runtime.h>
﻿extern "C" {
    __global__ void Run(int n, double* __restrict left, double* __restrict right, double* __restrict output) {
		int i = blockIdx.x*blockDim.x + threadIdx.x;
		if (i < n) output[i] = pow(left[i], right[i]);
	}
}